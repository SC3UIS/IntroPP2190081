
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

// Global variables used to define number of iterations and matrix values
#define T 10
#define MAX_VAL 1000
#define MIN_VAL 1
#define MAX_DIM 2000

__global__ void matrixMulGPU( int * a, int * b, int * c , int dim)
{
  int val = 0;

  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < dim && col < dim)
  {
    for ( int k = 0; k < dim; ++k )
      val += a[row * dim + k] * b[k * dim + col];
    c[row * dim + col] = val;
  }
  
}

void matrixMulCPU( int * a, int * b, int * c, int dim)
{
  int val = 0;

  for( int row = 0; row < dim; ++row )
    for( int col = 0; col < dim; ++col )
    {
      val = 0;
      for ( int k = 0; k < dim; ++k )
        val += a[row * dim + k] * b[k * dim + col];
      c[row * dim + col] = val;
    }
}

int main()
{
  // Structure to measure elapsed time during execution
  struct timeval t0, t1;
  
  int *a, *b, *c_cpu, *c_gpu, dim;

  int size = MAX_DIM * MAX_DIM * sizeof (int); // Number of bytes of an N x N matrix

  // Allocating memory using CUDA directives
  hipMallocManaged (&a, size);
  hipMallocManaged (&b, size);
  hipMallocManaged (&c_cpu, size);
  hipMallocManaged (&c_gpu, size);
  
  printf("--------------------------------------------\n");
  printf("  CUDA Matrix Multiplication Parallelizing\n");
  printf("--------------------------------------------\n");

  printf("\n");
  printf("--------------------------------------------\n");
  printf("    Size:  Sequential   vs   Parallel \n");
  printf("--------------------------------------------\n");
  
  /*
    There will be a total of T Matrix Multiplications
    its dimension will be increasing in ith powers of 2
    until the Tth power
    
    Times of execution will be measured and displayed
    if and only if there are no discrepancies in the
    resulting matrix
   */
  for (int i = 1; i <= T; i++){
    dim = pow(2, i);
    
    srandom(time(0)+clock()+random());
    for( int row = 0; row < dim; ++row )
      for( int col = 0; col < dim; ++col )
      {
        a[row*dim + col] = rand() % MAX_VAL + MIN_VAL;
        b[row*dim + col] = rand() % MAX_VAL + MIN_VAL;
        c_cpu[row*dim + col] = 0;
        c_gpu[row*dim + col] = 0;
      }
    
    // A 16 x 16 block threads
    dim3 threads_per_block (16, 16, 1); 
    dim3 number_of_blocks ((dim / threads_per_block.x) + 1, (dim / threads_per_block.y) + 1, 1);

    gettimeofday(&t0, 0);
    
    matrixMulGPU <<< number_of_blocks, threads_per_block >>> ( a, b, c_gpu, dim );

    // Directive used to wait until the asynchronous execution finishes, treating it as synchronous
    hipDeviceSynchronize(); 
    
    gettimeofday(&t1, 0);
    double res_gpu = (t1.tv_sec-t0.tv_sec) * 1.0f + (t1.tv_usec - t0.tv_usec) / 1000000.0f;
    
    gettimeofday(&t0, 0);
    // Call the CPU version to check our work
    matrixMulCPU( a, b, c_cpu, dim );
    
    gettimeofday(&t1, 0);
    double res_cpu = (t1.tv_sec-t0.tv_sec) * 1.0f + (t1.tv_usec - t0.tv_usec) / 1000000.0f;
    
    // Compare the two answers to make sure they are equal
    bool error = false;
    for( int row = 0; row < dim && !error; ++row )
      for( int col = 0; col < dim && !error; ++col )
        if (c_cpu[row * dim + col] != c_gpu[row * dim + col])
      {
          printf("FOUND ERROR at c[%d][%d]\n", row, col);
          error = true;
          break;
        }
    if (!error)
      printf("%d:    %f                    %f\n",dim, res_cpu, res_gpu  );
        
  }

  // Free all our allocated memory 
  hipFree(a); hipFree(b);
  hipFree( c_cpu ); hipFree( c_gpu );
  
  
}
